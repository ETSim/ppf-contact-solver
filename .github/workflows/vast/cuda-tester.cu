#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 1024; // Vector size
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc A");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc B");
    checkCudaError(hipMalloc(&d_C, size), "hipMalloc C");

    checkCudaError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice),
                   "hipMemcpy A");
    checkCudaError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice),
                   "hipMemcpy B");

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel synchronization");

    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost),
                   "hipMemcpy result");

    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            printf("Verification failed at index %d: %f != %f + %f\n", i,
                   h_C[i], h_A[i], h_B[i]);
            success = false;
            break;
        }
    }
    checkCudaError(hipFree(d_A), "hipFree A");
    checkCudaError(hipFree(d_B), "hipFree B");
    checkCudaError(hipFree(d_C), "hipFree C");
    free(h_A);
    free(h_B);
    free(h_C);

    if (success) {
        printf("CUDA test passed successfully!\n");
    }

    return 0;
}
